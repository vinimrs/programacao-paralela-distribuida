#include "hip/hip_runtime.h"
%%writefile cuda.cu
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>

// Solves the minimum distance between all pairs of vertices
__global__ void md_all_pairs (uint32_t* dists, uint32_t k, uint32_t v) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < v && j < v) {
    uint32_t distanceThroughK = dists[i*v+k] + dists[k*v+j];

    // Checks for overflows with the UINT32_MAX
    if ((distanceThroughK >= dists[i*v+k])
          &&(distanceThroughK >= dists[k*v+j])
          &&(distanceThroughK < dists[i*v+j])
        ) {
      dists[i*v+j] = distanceThroughK;
    }
  }
}

/* Computes the average minimum distance between all pairs of vertices with a path connecting them
    with CUDA */
__global__ void amd_cuda (uint32_t* dists, uint32_t v, uint32_t* data) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t infinity = v*v;
  uint32_t val = 1;

  // We only consider if the vertices are different and there is a path
  if ((i != j) && (j < v && i < v) &&
      (dists[i*v+j] != 0) && (dists[i*v+j] < infinity)
      ) {
    atomicAdd(&data[0], dists[i*v+j]);
    atomicAdd(&data[1], val);
  }

}

/* Computes the average minimum distance between all pairs of vertices with a path connecting them */
void amd (uint32_t* dists, uint32_t v) {
  uint32_t i, j;
	uint32_t infinity = v*v;
	uint32_t smd = 0; 	//sum of minimum distances
	uint32_t paths = 0; //number of paths found
	uint32_t solution = 0;

  for (i = 0; i < v; ++i) {
    for (j = 0; j < v; ++j) {
  // We only consider if the vertices are different and there is a path
      if ((i != j) && (dists[i*v+j] < infinity)) {
        smd += dists[i*v+j];
        paths++;
      }
    }
  }

	solution = smd / paths;
	printf("%d\n", solution);

}


/* Debug function (not to be used when measuring performance)*/
void debug (uint32_t* dists, uint32_t v) {
  uint32_t i, j;
	uint32_t infinity = v*v;

  for (i = 0; i < v; ++i) {
    for (j = 0; j < v; ++j) {
      if (dists[i*v+j] > infinity) printf("%7s", "inf");
      else printf ("%7u", dists[i*v+j]);
    }
    printf("\n");
  }
}

// Main program - reads input, calls FW, shows output
int main (int argc, char* argv[]) {
  uint32_t* d_dists; // pointer to matrix from GPU

  //Reads input
  //First line: v (number of vertices) and e (number of edges)
  uint32_t v, e;
  scanf("%u %u", &v, &e);

  //Allocates distances matrix (w/ size v*v) i
  //and sets it with max distance and 0 for own vertex
  size_t size = v*v*sizeof(uint32_t);
  uint32_t* dists = (uint32_t *) malloc(size);
  memset(dists, UINT32_MAX, size);
  uint32_t i;
  for ( i = 0; i < v; ++i ) dists[i*v+i] = 0;

  //Reads edges from file and sets them in the distance matrix
  uint32_t source, dest, cost;
  for ( i = 0; i < e; ++i ){
    scanf("%u %u %u", &source, &dest, &cost);
    if (cost < dists[source*v+dest]) dists[source*v+dest] = cost;
  }

  // Define number of threads with the number of vertices;
  int num_threads = v > 32 ? 32 : v;
  printf("num_threads = %d\n", num_threads);

  // Malloc of space to matrix in GPU
  hipMalloc((void **)&d_dists, size);

  // Copy of data from dists to d_dists
  hipMemcpy(d_dists, dists, size, hipMemcpyHostToDevice);

  // Defining block and grid
  dim3 block;

  // Standard setup with 252 threads per block
  block.x = 16;
  block.y = 16;
  block.z = 1;

  dim3 grid;
  grid.x = (v + block.x -1) / block.x; // ceil of division
  grid.y = (v + block.y -1) / block.y;
  grid.z = 1;

  //Computes the minimum distance for all pairs of vertices
  for(int k = 0; k < v; ++k) {
    md_all_pairs<<< grid, block >>>(d_dists, k, v);
  }

  // Copy result from gpu to cpu
  hipMemcpy(dists, d_dists, size, hipMemcpyDeviceToHost);

#if AMD_PAR == 1
  printf("Computing AMD parallely\n");
  // Computing the final solution
  uint32_t* d_data; // pointer to solution data on GPU
  size_t sizeData = 2 * sizeof(uint32_t);
  uint32_t* data = (uint32_t *) malloc(sizeData);
  data[0] = 0; // sum of minimum distances
  data[1] = 0; // number of paths found

  uint32_t solution = 0;

  hipMalloc((void **)&d_data, sizeData);

  // Copy of data from data to d_data
  hipMemcpy(d_data, data, sizeData, hipMemcpyHostToDevice);

  //Computes the final solution
  amd_cuda<<< grid, block >>>(d_dists, v, d_data);

  hipMemcpy(data, d_data, sizeData, hipMemcpyDeviceToHost);

  solution = data[0] / data[1];
  printf("%d\n", solution);

  hipFree(d_data); 
#else 
  printf("Computing AMD sequentially\n");
  //Computes and prints the final solution
  amd(dists, v);
#endif

#if DEBUG
	debug(dists, v);
#endif

  hipFree(d_dists);

  return 0;
}